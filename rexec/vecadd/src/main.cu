
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vec_add(int *a, int *b, int *c) {
    int tid = threadIdx.x;
    c[tid] = a[tid] + b[tid];
}

int main() {
    int n = 8;

    int *a_h, *b_h, *c_h;
    a_h = (int *) malloc(sizeof(int)*n);
    b_h = (int *) malloc(sizeof(int)*n);
    c_h = (int *) malloc(sizeof(int)*n);

    for (int i = 0; i < n; i++) {
        a_h[i] = i;
        b_h[i] = i;
    }

    int *a_d, *b_d, *c_d;
    hipMalloc((void **)&a_d, sizeof(int)*n);
    hipMalloc((void **)&b_d, sizeof(int)*n);
    hipMalloc((void **)&c_d, sizeof(int)*n);

    hipMemcpy(a_d, a_h, sizeof(int)*n, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, sizeof(int)*n, hipMemcpyHostToDevice);

    vec_add<<<1, n>>>(a_d, b_d, c_d);
    hipDeviceSynchronize();

    hipMemcpy(c_h, c_d, sizeof(int)*n, hipMemcpyDeviceToHost);
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    for (int i = 0; i < n; i++) {
        printf("v%d: %d\n", i, c_h[i]);
    }

    return 0;
}